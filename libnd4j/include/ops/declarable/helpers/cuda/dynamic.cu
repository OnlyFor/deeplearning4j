#include "hip/hip_runtime.h"
/* ******************************************************************************
 *
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 *  See the NOTICE file distributed with this work for additional
 *  information regarding copyright ownership.
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author raver119@gmail.com
//
#include <helpers/ConstantTadHelper.h>
#include <helpers/PointersManager.h>
#include <ops/declarable/helpers/dynamic.h>

#include "execution/cuda/LaunchDims.h"

namespace sd {
namespace ops {
namespace helpers {

template <typename X, typename Y>
static SD_KERNEL void dynamicPartitionScalarKernel(const void *vx, const sd::LongType *xShapeInfo, const void *vi,
                                                   const sd::LongType *iShapeInfo, void **vz,
                                                   sd::LongType **zShapeInfos, const sd::LongType numOutputs) {
  auto x = reinterpret_cast<const X *>(vx);
  auto i = reinterpret_cast<const Y *>(vi);
  auto xLength = shape::length(xShapeInfo);
  auto iLength = shape::length(iShapeInfo);

  extern __shared__ char shmem[];
  __shared__ Y *rawIndices;
  __shared__ Y *trueIndices;

  if (threadIdx.x == 0) {
    rawIndices = reinterpret_cast<Y *>(shmem);
    trueIndices = rawIndices + blockDim.x;
  }
  __syncthreads();

  // we run things in blocks, 1 partition per block of threads
  for (sd::LongType o = blockIdx.x; o < numOutputs; o += gridDim.x) {
    auto z = reinterpret_cast<X *>(vz[o]);

    auto zShapeInfo = zShapeInfos[o];
    auto zLength = shape::length(zShapeInfo);

    // iLimit should be multiple of blockDim.x
    auto iLimit = iLength <= blockDim.x ? blockDim.x : (iLength + (blockDim.x - (iLength % blockDim.x)));
    int cnt = 0;

    for (sd::LongType e = threadIdx.x; e < iLimit; e += blockDim.x) {
      // load set of indices into shared memory
      if (e < iLength) rawIndices[threadIdx.x] = i[shape::getIndexOffset(e, iShapeInfo)];
      __syncthreads();

      // now we need to find out where our actual updates will be mapped
      // TODO: this can be improved obviously, by using prefix-sum like approach
      if (threadIdx.x == 0) {
        for (int f = 0; f < blockDim.x; f++) {
          if (rawIndices[f] == static_cast<Y>(o))
            trueIndices[f] = cnt++;
          else
            trueIndices[f] = -1;
        }
      }
      __syncthreads();

      // doing actual update
      if (e < iLength)
        if (trueIndices[threadIdx.x] >= 0) {
          z[trueIndices[threadIdx.x]] = x[shape::getIndexOffset(e, xShapeInfo)];
        }

      __syncthreads();
    }
  }
}

template <typename X, typename Y>
static SD_KERNEL void dynamicPartitionTadKernel(const void *vx, const sd::LongType *xTadShapeInfo,
                                                const sd::LongType *xTadOffsets, sd::LongType xLength,
                                                const void *vindices, const sd::LongType *iShapeInfo,
                                                sd::LongType iLength, void **vz, sd::LongType **zTadShapeInfos,
                                                sd::LongType **zTadOffsets, sd::LongType numOutputs) {
  auto x = reinterpret_cast<const X *>(vx);
  auto indices = reinterpret_cast<const Y *>(vindices);

  // we run things in blocks, 1 partition per block of threads
  for (int i = blockIdx.x; i < numOutputs; i += gridDim.x) {
    auto z = reinterpret_cast<X *>(vz[i]);

    // each thread has own counter for partitions
    int outCnt = 0;

    for (sd::LongType e = 0; e < iLength; e++) {
      if (indices[shape::getIndexOffset(e, iShapeInfo)] == i) {
        auto dx = x + xTadOffsets[e];
        auto dz = z + zTadOffsets[i][outCnt++];

        for (int f = threadIdx.x; f < xLength; f += blockDim.x) {
          dz[shape::getIndexOffset(f, zTadShapeInfos[i])] = dx[shape::getIndexOffset(f, xTadShapeInfo)];
        }
      }
    }
  }
}

template <typename X, typename Y>
static void _dynamicPartitionFunctor(sd::LaunchContext *context, NDArray const *input, NDArray const *indices,
                                     std::vector<NDArray *> &outputList) {
  std::vector<std::pair<NDArray *, int>> outputs(outputList.size());
  int sourceDimsLen = input->rankOf() - indices->rankOf();

  unsigned int outSize = outputList.size();

  PointersManager pm(context, "dynamicPartition");

  if (sourceDimsLen) {  // non-linear case
    std::vector<sd::LongType> sourceDims(sourceDimsLen);

    for (int i = sourceDimsLen; i > 0; i--) sourceDims[sourceDimsLen - i] = input->rankOf() - i;
    // compute tad array for given dimensions
    auto packX = ConstantTadHelper::getInstance().tadForDimensions(input->shapeInfo(), &sourceDims);

    std::vector<void *> outBuffers(outSize);
    std::vector<const sd::LongType *> tadShapes(outSize);
    std::vector<const sd::LongType *> tadOffsets(outSize);
    std::vector<sd::LongType> numTads(outSize);
    // fill up dimensions array for before kernel
    for (unsigned int i = 0; i < outSize; i++) {
      outputs[i].first = outputList[i];
      std::vector<sd::LongType> outDims(outputs[i].first->rankOf() - 1);

      int r = outputs[i].first->rankOf();

      for (int k = 1; k < r; k++) outDims[k - 1] = k;

      auto packZ = ConstantTadHelper::getInstance().tadForDimensions(outputList.at(i)->shapeInfo(), &outDims);

      outBuffers[i] = outputList.at(i)->specialBuffer();
      tadShapes[i] = packZ->platformShapeInfo();
      tadOffsets[i] = packZ->platformOffsets();
    }

    // we copy pointers to device
    auto dOutBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void *)));
    auto dOutTadShapes = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(tadShapes.data(), tadShapes.size() * sizeof(sd::LongType *)));
    auto dOutTadOffsets = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(tadOffsets.data(), tadOffsets.size() * sizeof(sd::LongType *)));
    // run kernel on device
    dim3 launchDims = getDynamicPartitionDims(256,sizeof(Y));

    dynamicPartitionTadKernel<X, Y><<<launchDims.y,launchDims.x, launchDims.z, *context->getCudaStream()>>>(
        input->specialBuffer(), packX->platformShapeInfo(), packX->platformOffsets(),
        shape::length(packX->primaryShapeInfo()), indices->specialBuffer(), indices->specialShapeInfo(),
        indices->lengthOf(), dOutBuffers, dOutTadShapes, dOutTadOffsets, outSize);

  } else {  // linear case
    dim3 launchDims = getDynamicPartitionDims(256,sizeof(Y));
    std::vector<void *> outBuffers;
    std::vector<const sd::LongType *> outShapes;

    for (auto v : outputList) {
      outBuffers.emplace_back(v->specialBuffer());
      outShapes.emplace_back(v->specialShapeInfo());
    }

    auto dOutBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(outBuffers.data(), outBuffers.size() * sizeof(void *)));
    auto dOutShapes = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(outShapes.data(), outShapes.size() * sizeof(sd::LongType *)));

    dynamicPartitionScalarKernel<X, Y><<<launchDims.y,launchDims.x, launchDims.z, *context->getCudaStream()>>>(
        input->specialBuffer(), input->specialShapeInfo(), indices->specialBuffer(), indices->specialShapeInfo(),
        dOutBuffers, dOutShapes, outSize);
  }

  pm.synchronize();
}

template <typename X, typename Y>
static SD_KERNEL void dynamicStitchScalarKernel(void **vx, sd::LongType **xShapeInfos, void **vindices,
                                                sd::LongType **iShapeInfos, int inputSize, void *vz,
                                                const sd::LongType *zShapeInfo, sd::LongType zLength) {
  auto z = reinterpret_cast<X *>(vz);

  for (int e = blockIdx.x; e < inputSize; e += gridDim.x) {
    auto x = reinterpret_cast<X *>(vx[e]);
    auto indices = reinterpret_cast<Y *>(vindices[e]);

    auto xShapeInfo = xShapeInfos[e];
    auto iShapeInfo = iShapeInfos[e];

    auto iLength = shape::length(iShapeInfo);

    for (int i = threadIdx.x; i < iLength; i += blockDim.x) {
      auto idx = indices[shape::getIndexOffset(i, iShapeInfo)];
      if (idx >= 0 && idx < zLength)
        z[shape::getIndexOffset(idx, zShapeInfo)] = x[shape::getIndexOffset(i, xShapeInfo)];
    }
  }
}

template <typename X, typename Y>
static SD_KERNEL void dynamicStitchTadKernel(void **vx, sd::LongType **xTadShapeInfos, sd::LongType **xTadOffsets,
                                             void **vindices, sd::LongType **iShapeInfos, int inputSize, void *vz,
                                             const sd::LongType *zTadShapeInfo, const sd::LongType *zTadOffsets,
                                             sd::LongType *numTadsPerInput, sd::LongType numOutputsTad) {
  //note: this implementation is less than ideal but several forms of parallelization do not seem to work.
  //for now since this isn't a computationally intensive function this serial implementation that works correctly
  //will stay.
  auto bz = reinterpret_cast<X *>(vz);
  int arrIndex = threadIdx.x;
  //each input
  for (int e = arrIndex; e < inputSize; e++) {
    auto indices = reinterpret_cast<Y *>(vindices[e]);

    auto iShapeInfo = iShapeInfos[e];
    auto numTads = numTadsPerInput[e];
    if (shape::isEmpty(iShapeInfo)) continue;

    auto iLength = shape::length(iShapeInfo);
    auto zLength = shape::length(zTadShapeInfo);

    auto xTadShapeInfo = xTadShapeInfos[e];
    auto xTadLength = shape::length(xTadShapeInfo);

    // process each index setting values for this tad
    for (int i = 0; i < iLength; i++) {
      auto idx = indices[shape::getIndexOffset(i, iShapeInfo)];

      // the input at a given index starting at the offset for the current tad
      auto x = reinterpret_cast<X *>(vx[e]) + xTadOffsets[e][i];
      auto zTad = bz + zTadOffsets[idx];
      for (int j = 0; j < xTadLength; j++) {
        auto xIdx = shape::getIndexOffset(j, xTadShapeInfo);
        auto zIdx = shape::getIndexOffset(j, zTadShapeInfo);
        if (xIdx < xTadLength && xIdx >= 0 && zIdx < zLength && zIdx >= 0) zTad[zIdx] = x[xIdx];
      }
    }
  }

  __syncthreads();

}


template <typename X, typename Y>
static sd::Status _dynamicStitchFunctor(sd::LaunchContext *context, std::vector<NDArray *> const &inputs,
                                        std::vector<NDArray *> const &indices, NDArray *output) {
  sd::LongType inputSize = inputs.size();

  PointersManager pm(context, "dynamicStitch");

  if (output->isVector()) {
    std::vector<const void *> inputBuffers(inputSize);
    std::vector<const sd::LongType *> inputShapes(inputSize);
    std::vector<const void *> indicesBuffers(inputSize);
    std::vector<const sd::LongType *> indicesShapes(inputSize);

    for (sd::LongType e = 0; e < inputSize; e++) {
      inputBuffers[e] = inputs.at(e)->specialBuffer();
      indicesBuffers[e] = indices.at(e)->specialBuffer();

      inputShapes[e] = inputs.at(e)->specialShapeInfo();
      indicesShapes[e] = indices.at(e)->specialShapeInfo();
    }

    // copying pointers to buffers to device
    auto dInputBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(inputBuffers.data(), inputSize * sizeof(void *)));
    auto dIndicesBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(indicesBuffers.data(), inputSize * sizeof(void *)));
    auto dInputShapes =
        reinterpret_cast<sd::LongType **>(pm.replicatePointer(inputShapes.data(), inputSize * sizeof(sd::LongType *)));
    auto dIndicesShapes = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(indicesShapes.data(), inputSize * sizeof(sd::LongType *)));
    dim3 launchDims = getLaunchDims("dynamic_stitch_tad");

    dynamicStitchScalarKernel<X, Y><<<launchDims.y, launchDims.x, launchDims.z, *context->getCudaStream()>>>(
        dInputBuffers, dInputShapes, dIndicesBuffers, dIndicesShapes, inputSize, output->specialBuffer(),
        output->specialShapeInfo(), output->lengthOf());
  } else {
    std::vector<sd::LongType> restDims(output->rankOf() - 1);
    for (int i = restDims.size(); i > 0; i--) restDims[restDims.size() - i] = output->rankOf() - i;
    //print dims:
    printf("rest dims for output\n");
    for(int i = 0; i < restDims.size(); i++) {
      printf("%d ",restDims[i]);
    }
    printf("\n");

    auto packZ = ConstantTadHelper::getInstance().tadForDimensions(output->shapeInfo(), &restDims);

    std::vector<const void *> inputBuffers(inputSize);
    std::vector<const sd::LongType *> inputTadShapes(inputSize);
    std::vector<const sd::LongType *> inputTadOffsets(inputSize);

    std::vector<const void *> indicesBuffers(inputSize);
    std::vector<const sd::LongType *> indicesShapes(inputSize);
    std::vector<sd::LongType > inputsNumTads(inputSize);

    for (sd::LongType e = 0; e < inputSize; e++) {
      std::vector<sd::LongType> sourceDims(inputs[e]->rankOf() - indices[e]->rankOf());
      for (sd::LongType  i = sourceDims.size(); i > 0; i--) sourceDims[sourceDims.size() - i] = inputs[e]->rankOf() - i;

      auto packX = ConstantTadHelper::getInstance().tadForDimensions(inputs[e]->shapeInfo(), &sourceDims);
      printf("tad shape info for input %d\n",e);
      shape::printShapeInfo(packX->primaryShapeInfo());
      indicesBuffers[e] = indices[e]->specialBuffer();
      indicesShapes[e] = indices[e]->specialShapeInfo();
      inputsNumTads[e] = packX->numberOfTads();
      inputBuffers[e] = inputs[e]->specialBuffer();
      inputTadShapes[e] = packX->platformShapeInfo();
      inputTadOffsets[e] = packX->platformOffsets();
    }

    // copying pointers to buffers to device
    auto dInputBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(inputBuffers.data(), inputSize * sizeof(void *)));
    auto dInputTadShapes = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(inputTadShapes.data(), inputSize * sizeof(sd::LongType *)));
    auto dInputTadOffsets = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(inputTadOffsets.data(), inputSize * sizeof(sd::LongType *)));

    auto dIndicesBuffers =
        reinterpret_cast<void **>(pm.replicatePointer(indicesBuffers.data(), inputSize * sizeof(void *)));
    auto dIndicesShapes = reinterpret_cast<sd::LongType **>(
        pm.replicatePointer(indicesShapes.data(), inputSize * sizeof(sd::LongType *)));

    auto dNumTadsInputs = reinterpret_cast<sd::LongType *>(
        pm.replicatePointer(inputsNumTads.data(), inputSize * sizeof(sd::LongType *)));


    dim3 launchDims = getLaunchDims("dynamic_stitch_tad");
    printf("dynamic stitch tad dimensions: %d %d %d\n", launchDims.x, launchDims.y, launchDims.z);
    dynamicStitchTadKernel<X, Y><<<launchDims.x, launchDims.y, launchDims.z, *context->getCudaStream()>>>(
        dInputBuffers, dInputTadShapes, dInputTadOffsets, dIndicesBuffers, dIndicesShapes, inputSize,
        output->specialBuffer(), packZ->platformShapeInfo(), packZ->platformOffsets(),dNumTadsInputs, packZ->numberOfTads());
  }

  pm.synchronize();

  return sd::Status::OK;
}

template <typename T>
static void _dynamicPartitionFunctorBP(NDArray const *input, NDArray const *indices,
                                       std::vector<NDArray *> const &inputGradientList,
                                       std::vector<NDArray *> &outputList) {}

void dynamicPartitionFunctor(sd::LaunchContext *context, NDArray const *input, NDArray const *indices,
                             std::vector<NDArray *> &outputList) {
  auto xType = input->dataType();
  auto yType = indices->dataType();

  NDArray::prepareSpecialUse({}, {indices, input});

  BUILD_DOUBLE_SELECTOR(xType, yType, _dynamicPartitionFunctor, (context, input, indices, outputList), SD_NUMERIC_TYPES,
                        SD_INDEXING_TYPES);

  NDArray::registerSpecialUse({}, {indices, input});

  // TODO: it would be nice to have NDArray::registerSpecialUse signature that accepts something else beyond
  // initializer_list
  for (auto v : outputList) {
    v->tickWriteDevice();
  }
}

template <typename T>
static sd::Status _dynamicStitchFunctorBP(std::vector<NDArray *> const &inputs, std::vector<NDArray *> const &indices,
                                          NDArray const *gradInput, std::vector<NDArray *> &outputList) {
  THROW_EXCEPTION("Not implemented yet");
}

sd::Status dynamicStitchFunctor(sd::LaunchContext *context, std::vector<NDArray *> const &inputs,
                                std::vector<NDArray *> const &indices, NDArray *output) {
  auto xType = inputs.at(0)->dataType();
  auto yType = indices.at(0)->dataType();

  for (auto v : indices) {
    v->syncToDevice();
    v->tickReadDevice();
  }

  for (auto v : inputs) {
    v->syncToDevice();
    v->tickReadDevice();
  }

  NDArray::prepareSpecialUse({output}, {});

  BUILD_DOUBLE_SELECTOR(xType, yType, _dynamicStitchFunctor, (context, inputs, indices, output), SD_NUMERIC_TYPES,
                        SD_INDEXING_TYPES);

  NDArray::registerSpecialUse({output}, {});

  return sd::Status::OK;
}

sd::Status dynamicStitchFunctorBP(sd::LaunchContext *context, std::vector<NDArray *> const &inputs,
                                  std::vector<NDArray *> const &indices, NDArray const *gradInput,
                                  std::vector<NDArray *> &outputList) {
  auto xType = inputs.at(0)->dataType();

  BUILD_SINGLE_SELECTOR(xType, return _dynamicStitchFunctorBP, (inputs, indices, gradInput, outputList),
                        SD_NUMERIC_TYPES);
}

void dynamicPartitionFunctorBP(sd::LaunchContext *context, NDArray const *input, NDArray const *indices,
                               std::vector<NDArray *> const &inputGradientList, std::vector<NDArray *> &outputList) {
  auto xType = input->dataType();

  BUILD_SINGLE_SELECTOR(xType, _dynamicPartitionFunctorBP, (input, indices, inputGradientList, outputList),
                        SD_NUMERIC_TYPES);
}

}  // namespace helpers
}  // namespace ops
}  // namespace sd
